#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "sum_array_device.cuh"

__global__
void naiveSumArray(const float *input, float *output, int n) {
    float partial_sum = 0.f;
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        float val = input[i];
        partial_sum += val;
    }
    atomicAdd(output, partial_sum);
}

void cudaSumArray(
    const float *d_input,
    float *d_output,
    int n,
    SumArrayImplementation type)
{
    if (type == NAIVE) {
        dim3 blockSize(1024, 1);
        dim3 gridSize(1, 1);
        naiveSumArray<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
}