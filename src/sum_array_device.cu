#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "sum_array_device.cuh"

__global__
void naiveSumArray(const float *input, float *output, int n) {
    double sum = 0.0;
    //reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < n; 
            i += blockDim.x * gridDim.x) {
        sum += input[i];
    }
    atomicAdd(output, sum);
}


void cudaSumArray(
    const float *d_input,
    float *d_output,
    int n,
    SumArrayImplementation type);
{
    if (type == NAIVE) {
        dim3 blockSize(1024, 1);
        dim3 gridSize(n / 1024, 1);
        naiveSumArray<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
}