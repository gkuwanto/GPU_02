#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "sum_array_device.cuh"

__global__
void naiveSumArray(const float *input, float *output, int n) {
    float partial_sum = 0.0;
    //reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < n; 
            i += blockDim.x * gridDim.x) {
        partial_sum += input[i];
    }
    atomicAdd(output, partial_sum);
}


void cudaSumArray(
    const float *d_input,
    float *d_output,
    int n,
    SumArrayImplementation type)
{
    if (type == NAIVE) {
        dim3 blockSize(1024, 1);
        dim3 gridSize(n / 32 / 1024, 1);
        naiveSumArray<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
}