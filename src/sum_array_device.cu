#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "sum_array_device.cuh"

__global__
void naiveSumArray(const float *input, float *output, int n) {
    double sum = input[blockIdx.x * blockDim.x + threadIdx.x];
    atomicAdd(output, sum);
}


void cudaSumArray(
    const float *d_input,
    float *d_output,
    int n,
    SumArrayImplementation type)
{
    if (type == NAIVE) {
        dim3 blockSize(1024, 1);
        dim3 gridSize(n / 1024, 1);
        naiveSumArray<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
}